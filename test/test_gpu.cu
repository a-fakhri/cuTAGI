#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         test_gpu.cu
// Description:  Main script to test the GPU implementation of cuTAGI
// Authors:      Florensa, Miquel, Luong-Ha Nguyen & James-A. Goulet
// Created:      February 20, 2023
// Updated:      April 13, 2023
// Contact:      miquelflorensa11@gmail.com, luongha.nguyen@gmail.com &
//               james.goulet@polymtl.ca
// Copyright (c) 2023 Miquel Florensa, Luong-Ha Nguyen & James-A. Goulet.
// Some rights reserved.
///////////////////////////////////////////////////////////////////////////////

#include "test_gpu.cuh"

const int NUM_TESTS = 9;

int test_gpu(std::vector<std::string>& user_input_options,
             int num_tests_passed_cpu) {
    std::string reinizialize_test_outputs = "";
    std::string test_architecture = "";
    std::string date = "";
    bool single_test = true;

    if (user_input_options.size() == 1 &&
        (user_input_options[0] == "-h" || user_input_options[0] == "--help")) {
        // Help message have already been showed in test_cpu.cpp
        return -1;
    } else if (user_input_options.size() > 0 && user_input_options.size() < 3) {
        if (user_input_options[0] == "-reset") {
            if (user_input_options.size() == 1) {
                reinizialize_test_outputs = "all";
            } else {
                // Check if the architecture is valid
                check_valid_input_architecture(user_input_options[1]);

                reinizialize_test_outputs = user_input_options[1];
            }
        } else {
            // Check if the architecture is valid
            check_valid_input_architecture(user_input_options[0]);

            test_architecture = user_input_options[0];
        }
        std::time_t t = std::time(0);  // get time now
        std::tm* now = std::localtime(&t);
        std::string year = std::to_string(now->tm_year + 1900);
        std::string month = std::to_string(now->tm_mon + 1);
        if (month.size() == 1) month = "0" + month;
        std::string day = std::to_string(now->tm_mday);
        if (day.size() == 1) day = "0" + day;

        date = year + "_" + month + "_" + day;

    } else if (user_input_options.size() == 0) {
        test_architecture = "all";
        single_test = false;
    } else if (user_input_options.size() > 1) {
        std::cout << "Too many arguments" << std::endl;
        return -1;
    }

    // Read last test dates
    std::vector<std::string> test_dates = read_dates();

    // Index of the current test
    int test_num;

    ////////////////////////////
    //      PERFORM TESTS     //
    ////////////////////////////

    if (test_architecture.size() > 0) {
        int num_test_passed = num_tests_passed_cpu;

        // Perform test on GPU for the classification task
        if (test_architecture == "all" || test_architecture == "cnn") {
            test_num = 6;  // CNN

            bool test_result =
                test_cnn_gpu(false, test_dates[test_num], "cnn", "mnist");

            print_test_results(single_test, test_result, NUM_TESTS, test_num,
                               "CNN");

            if (test_result) num_test_passed++;
        }

        // Perform test on GPU with batch normalization for classification task
        if (test_architecture == "all" ||
            test_architecture == "cnn_batch_norm") {
            test_num = 7;  // CNN batch norm.

            bool test_result = test_cnn_batch_norm_gpu(
                false, test_dates[test_num], "cnn_batch_norm", "mnist");

            print_test_results(single_test, test_result, NUM_TESTS, test_num,
                               "CNN batch normalization");

            if (test_result) num_test_passed++;
        }

        // Perform test on GPU with autoencoder for image generation
        if (test_architecture == "all" || test_architecture == "autoencoder") {
            test_num = 8;  // Autoencoder

            bool test_result = test_autoencoder_gpu(false, test_dates[test_num],
                                                    "autoencoder", "mnist");

            print_test_results(single_test, test_result, NUM_TESTS, test_num,
                               "Autoencoder");

            if (test_result) num_test_passed++;
        }

        // Number of tests passed
        if (test_architecture == "all") {
            std::cout << std::endl;
            std::cout << "--------------------SUMMARY--------------------"
                      << std::endl;
            std::cout << "Passed tests: [" << num_test_passed << "/"
                      << NUM_TESTS << "]" << std::endl;
            return num_test_passed;
        }
        return -1;
    }

    ///////////////////////////////
    // REINIZIALIZE TEST OUTPUTS //
    ///////////////////////////////

    if (reinizialize_test_outputs.size() > 0) {
        if (reinizialize_test_outputs == "all" ||
            reinizialize_test_outputs == "cnn") {
            // Reinizialize test outputs for classification task
            std::cout << "Reinizializing CNN test outputs" << std::endl;

            test_cnn_gpu(true, date, "cnn", "mnist");

            test_num = 6;  // CNN

            // Update de last date of the test
            write_dates(test_dates, test_num, date);
            test_dates[test_num] = date;
        }

        if (reinizialize_test_outputs == "all" ||
            reinizialize_test_outputs == "cnn_batch_norm") {
            // Reinizialize test outputs for classification task
            std::cout << "Reinizializing CNN batch norm. test outputs"
                      << std::endl;

            test_cnn_batch_norm_gpu(true, date, "cnn_batch_norm", "mnist");

            test_num = 7;  // CNN

            // Update de last date of the test
            write_dates(test_dates, test_num, date);
            test_dates[test_num] = date;
        }

        if (reinizialize_test_outputs == "all" ||
            reinizialize_test_outputs == "autoencoder") {
            // Reinizialize test outputs for for image generation
            std::cout << "Reinizializing Autoencoder test outputs" << std::endl;

            test_autoencoder_gpu(true, date, "autoencoder", "mnist");

            test_num = 8;  // Autoencoder

            // Update de last date of the test
            write_dates(test_dates, test_num, date);
            test_dates[test_num] = date;
        }
        return 0;
    }

    return -1;
}
