#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// File:         lstm_feed_backward.cu
// Description:  Long-Short Term Memory (LSTM) state backward pass in TAGI
// Authors:      Luong-Ha Nguyen & James-A. Goulet
// Created:      August 07, 2022
// Updated:      September 11, 2022
// Contact:      luongha.nguyen@gmail.com & james.goulet@polymtl.ca
// Copyright (c) 2022 Luong-Ha Nguyen & James-A. Goulet. Some rights reserved.
////////////////////////////////////////////////////////////////////////////////

#include "../include/lstm_feed_backward.cuh"

__global__ void lstm_delta_mean_var_z(
    float const *Sz, float const *mw, float const *Jf_ga, float const *mi_ga,
    float const *Ji_ga, float const *mc_ga, float const *Jc_ga,
    float const *mo_ga, float const *Jo_ga, float const *mc_prev,
    float const *mca, float const *Jca, float const *delta_m,
    float const *delta_S, int z_pos_i, int z_pos_o, int z_pos_o_lstm,
    int w_pos_f, int w_pos_i, int w_pos_c, int w_pos_o, int no, int ni,
    int seq_len, int B, float *delta_mz, float *delta_Sz)
/*Compute the updated quatitites of the mean of the hidden states for lstm
   layer*/
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum_mf, sum_mi, sum_mc, sum_mo, sum_Sz;
    float Czz_f, Czz_i, Czz_c, Czz_o;
    int k, m, i, x, y;
    if (row < B * seq_len && col < ni) {
        x = row / seq_len;
        y = row % seq_len;

        sum_mf = 0;
        sum_mi = 0;
        sum_mc = 0;
        sum_mo = 0;
        sum_Sz = 0;
        for (int j = 0; j < no; j++) {
            k = j + x * no * seq_len + y * no + z_pos_o_lstm;
            i = j + x * no * seq_len + y * no + z_pos_o;

            // Forget gate
            Czz_f = Jca[k] * mo_ga[k] * Jf_ga[k] *
                    mw[(ni + no) * j + col + w_pos_f] * mc_prev[k];
            sum_mf += Czz_f * delta_m[i];

            // Input gate
            Czz_i = Jca[k] * mo_ga[k] * Ji_ga[k] *
                    mw[(ni + no) * j + col + w_pos_i] * mc_ga[k];
            sum_mi += Czz_i * delta_m[i];

            // Cell state gate
            Czz_c = Jca[k] * mo_ga[k] * Jc_ga[k] *
                    mw[(ni + no) * j + col + w_pos_c] * mi_ga[k];
            sum_mc += Czz_c * delta_m[i];

            // Output gate
            Czz_o = Jo_ga[k] * mw[(ni + no) * j + col + w_pos_o] * mca[k];
            sum_mo += Czz_o * delta_m[i];
            sum_Sz += powf(Czz_f + Czz_i + Czz_c + Czz_o, 2) * delta_S[i];
        }

        // Updating quantities
        m = x * ni * seq_len + y * ni + col;
        delta_mz[m] = (sum_mf + sum_mi + sum_mc + sum_mo) * Sz[m + z_pos_i];
        delta_Sz[m] = Sz[m + z_pos_i] * sum_Sz * Sz[m + z_pos_i];
    }
}

__global__ void lstm_delta_mean_var_w(
    float const *Sw, float const *mha, float const *Jf_ga, float const *mi_ga,
    float const *Ji_ga, float const *mc_ga, float const *Jc_ga,
    float const *mo_ga, float const *Jo_ga, float const *mc_prev,
    float const *mca, float const *Jc, float const *delta_m,
    float const *delta_S, int z_pos_o, int z_pos_o_lstm, int w_pos_f,
    int w_pos_i, int w_pos_c, int w_pos_o, int no, int ni, int seq_len, int B,
    float *delta_mw, float *delta_Sw)
/*Compute updating quantities of the weight parameters for lstm layer */
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum_mf, sum_Sf, Cwa_f, sum_mi, sum_Si, Cwa_i, sum_mc, sum_Sc, Cwa_c,
        sum_mo, sum_So, Cwa_o;
    int k, m, l, i, x, y;
    if (row < (ni + no) && col < no) {
        sum_mf = 0;
        sum_Sf = 0;
        sum_mi = 0;
        sum_Si = 0;
        sum_mc = 0;
        sum_Sc = 0;
        sum_mo = 0;
        sum_So = 0;
        for (int t = 0; t < B * seq_len; t++) {
            x = t / seq_len;
            y = t % seq_len;

            k = col + y * seq_len + no * seq_len * x + z_pos_o_lstm;
            i = col + y * seq_len + no * seq_len * x + z_pos_o;
            l = row + y * (ni + no) + (ni + no) * seq_len * x;

            // Forget gate
            Cwa_f = Jc[k] * Jf_ga[k] * mc_prev[k] * mo_ga[k] * mha[l];
            sum_mf += Cwa_f * delta_m[i];
            sum_Sf += Cwa_f * delta_S[i] * Cwa_f;

            // Input gate
            Cwa_i = Jc[k] * Ji_ga[k] * mc_ga[k] * mo_ga[k] * mha[l];
            sum_mi += Cwa_i * delta_m[i];
            sum_Si += Cwa_i * delta_S[i] * Cwa_i;

            // Cell state gate
            Cwa_c = Jc[k] * Jc_ga[k] * mi_ga[k] * mo_ga[k] * mha[l];
            sum_mc += Cwa_c * delta_m[i];
            sum_Sc += Cwa_c * delta_S[i] * Cwa_c;

            // Output gate
            Cwa_o = Jo_ga[k] * mca[k] * mha[l];
            sum_mo += Cwa_o * delta_m[i];
            sum_So += Cwa_o * delta_S[i] * Cwa_o;
        }
        // Updating quantities for weights
        m = col * (ni + no) + row;
        delta_mw[m + w_pos_f] = sum_mf * Sw[m + w_pos_f];
        delta_Sw[m + w_pos_f] = Sw[m + w_pos_f] * sum_Sf * Sw[m + w_pos_f];

        delta_mw[m + w_pos_i] = sum_mi * Sw[m + w_pos_i];
        delta_Sw[m + w_pos_i] = Sw[m + w_pos_i] * sum_Si * Sw[m + w_pos_i];

        delta_mw[m + w_pos_c] = sum_mc * Sw[m + w_pos_c];
        delta_Sw[m + w_pos_c] = Sw[m + w_pos_c] * sum_Sc * Sw[m + w_pos_c];

        delta_mw[m + w_pos_o] = sum_mo * Sw[m + w_pos_o];
        delta_Sw[m + w_pos_o] = Sw[m + w_pos_o] * sum_So * Sw[m + w_pos_o];
    }
}

__global__ void lstm_delta_mean_var_b(
    float const *Sb, float const *Jf_ga, float const *mi_ga, float const *Ji_ga,
    float const *mc_ga, float const *Jc_ga, float const *mo_ga,
    float const *Jo_ga, float const *mc_prev, float const *mca, float const *Jc,
    float const *delta_m, float const *delta_S, int z_pos_o, int z_pos_o_lstm,
    int b_pos_f, int b_pos_i, int b_pos_c, int b_pos_o, int no, int seq_len,
    int B, float *delta_mb, float *delta_Sb)
/*Compute updating quantities of the bias for the lstm layer */
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum_mf, sum_Sf, Cwa_f, sum_mi, sum_Si, Cwa_i, sum_mc, sum_Sc, Cwa_c,
        sum_mo, sum_So, Cwa_o;
    int k, i, x, y;
    if (col < no) {
        sum_mf = 0;
        sum_Sf = 0;
        sum_mi = 0;
        sum_Si = 0;
        sum_mc = 0;
        sum_Sc = 0;
        sum_mo = 0;
        sum_So = 0;
        for (int t = 0; t < B * seq_len; t++) {
            x = t / seq_len;
            y = t % seq_len;

            k = col + y * seq_len + no * seq_len * x + z_pos_o_lstm;
            i = col + y * seq_len + no * seq_len * x + z_pos_o;

            // Forget gate
            Cwa_f = Jc[k] * Jf_ga[k] * mc_prev[k] * mo_ga[k];
            sum_mf += Cwa_f * delta_m[i];
            sum_Sf += Cwa_f * delta_S[i] * Cwa_f;

            // Input gate
            Cwa_i = Jc[k] * Ji_ga[k] * mc_ga[k] * mo_ga[k];
            sum_mi += Cwa_i * delta_m[i];
            sum_Si += Cwa_i * delta_S[i] * Cwa_i;

            // Cell state gate
            Cwa_c = Jc[k] * Jc_ga[k] * mi_ga[k] * mo_ga[k];
            sum_mc += Cwa_c * delta_m[i];
            sum_Sc += Cwa_c * delta_S[i] * Cwa_c;

            // Output gate
            Cwa_o = Jo_ga[k] * mca[k];
            sum_mo += Cwa_o * delta_m[i];
            sum_So += Cwa_o * delta_S[i] * Cwa_o;
        }
        // Updating quantities for biases
        delta_mb[col + b_pos_f] = sum_mf * Sb[col + b_pos_f];
        delta_Sb[col + b_pos_f] =
            Sb[col + b_pos_f] * sum_Sf * Sb[col + b_pos_f];

        delta_mb[col + b_pos_i] = sum_mi * Sb[col + b_pos_i];
        delta_Sb[col + b_pos_i] =
            Sb[col + b_pos_i] * sum_Si * Sb[col + b_pos_i];

        delta_mb[col + b_pos_c] = sum_mc * Sb[col + b_pos_c];
        delta_Sb[col + b_pos_c] =
            Sb[col + b_pos_c] * sum_Sc * Sb[col + b_pos_c];

        delta_mb[col + b_pos_o] = sum_mo * Sb[col + b_pos_o];
        delta_Sb[col + b_pos_o] =
            Sb[col + b_pos_o] * sum_So * Sb[col + b_pos_o];
    }
}

void lstm_state_update(Network &net, StateGPU &state, ParamGPU &theta,
                       DeltaStateGPU &d_state, int l)
/*Update lstm's hidden states*/
{
    // Initialization
    int ni = net.nodes[l];
    int no = net.nodes[l + 1];
    int z_pos_i = net.z_pos[l];
    int z_pos_o = net.z_pos[l + 1];
    int z_pos_o_lstm = net.z_pos_lstm[l + 1];
    int w_pos_f, w_pos_i, w_pos_c, w_pos_o;
    int ni_c = ni + no;
    int b_seq = net.batch_size * net.input_seq_len;

    w_pos_f = net.w_pos[l];
    w_pos_i = net.w_pos[l] + ni_c * no;
    w_pos_c = net.w_pos[l] + 2 * ni_c * no;
    w_pos_o = net.w_pos[l] + 3 * ni_c * no;

    // Launch kernel
    int THREADS = net.num_gpu_threads;
    unsigned int gridRow_cov = (b_seq + THREADS - 1) / THREADS;
    unsigned int gridCol_cov = (ni + THREADS - 1) / THREADS;
    dim3 dimGrid_cov(gridCol_cov, gridRow_cov);
    dim3 dimBlock(THREADS, THREADS);

    lstm_delta_mean_var_z<<<dimGrid_cov, dimBlock>>>(
        state.d_Sz, theta.d_mw, state.lstm.d_Jf_ga, state.lstm.d_mi_ga,
        state.lstm.d_Ji_ga, state.lstm.d_mc_ga, state.lstm.d_Jc_ga,
        state.lstm.d_mo_ga, state.lstm.d_Jo_ga, state.lstm.d_mc_prev,
        state.lstm.d_mca, state.lstm.d_Jca, d_state.d_delta_m,
        d_state.d_delta_S, z_pos_i, z_pos_o, z_pos_o_lstm, w_pos_f, w_pos_i,
        w_pos_c, w_pos_o, no, ni, net.input_seq_len, net.batch_size,
        d_state.d_delta_mz, d_state.d_delta_Sz);
}

void lstm_parameter_update(Network &net, StateGPU &state, ParamGPU &theta,
                           DeltaStateGPU &d_state, DeltaParamGPU &d_theta,
                           int l)
/*Update lstm's parameters*/
{
    // Initialization
    int ni = net.nodes[l];
    int no = net.nodes[l + 1];
    int z_pos_i = net.z_pos[l];
    int z_pos_o = net.z_pos[l + 1];
    int z_pos_o_lstm = net.z_pos_lstm[l + 1];
    int w_pos_f, b_pos_f, w_pos_i, b_pos_i, w_pos_c, b_pos_c, w_pos_o, b_pos_o;
    int ni_c = ni + no;

    w_pos_f = net.w_pos[l];
    b_pos_f = net.b_pos[l];
    w_pos_i = net.w_pos[l] + ni_c * no;
    b_pos_i = net.b_pos[l] + no;
    w_pos_c = net.w_pos[l] + 2 * ni_c * no;
    b_pos_c = net.b_pos[l] + 2 * no;
    w_pos_o = net.w_pos[l] + 3 * ni_c * no;
    b_pos_o = net.b_pos[l] + 3 * no;

    // Launch kernel
    int THREADS = net.num_gpu_threads;
    unsigned int BLOCKS = (no + THREADS - 1) / THREADS;
    unsigned int gridRow = (ni + no + +THREADS - 1) / THREADS;
    unsigned int gridCol = (no + THREADS - 1) / THREADS;
    dim3 dimGrid(gridCol, gridRow);
    dim3 dimBlock(THREADS, THREADS);

    // Concatenate the hidden states from the previous time step and
    // activations from the previous layer
    unsigned int gridRow_cat = (net.batch_size + THREADS - 1) / THREADS;
    unsigned int gridCol_cat = (net.input_seq_len + THREADS - 1) / THREADS;
    dim3 dimGrid_cat(gridCol_cat, gridRow_cat);
    cat_activations_and_prev_states<<<dimGrid_cat, dimBlock>>>(
        state.d_ma, state.lstm.d_mh_prev, ni, no, net.input_seq_len,
        net.batch_size, z_pos_i, z_pos_o_lstm, state.lstm.d_mha);

    lstm_delta_mean_var_w<<<dimGrid, dimBlock>>>(
        theta.d_Sw, state.lstm.d_mha, state.lstm.d_Jf_ga, state.lstm.d_mi_ga,
        state.lstm.d_Ji_ga, state.lstm.d_mc_ga, state.lstm.d_Jc_ga,
        state.lstm.d_mo_ga, state.lstm.d_Jo_ga, state.lstm.d_mc_prev,
        state.lstm.d_mca, state.lstm.d_Jca, d_state.d_delta_m,
        d_state.d_delta_S, z_pos_o, z_pos_o_lstm, w_pos_f, w_pos_i, w_pos_c,
        w_pos_o, no, ni, net.input_seq_len, net.batch_size, d_theta.d_delta_mw,
        d_theta.d_delta_Sw);

    lstm_delta_mean_var_b<<<BLOCKS, THREADS>>>(
        theta.d_Sb, state.lstm.d_Jf_ga, state.lstm.d_mi_ga, state.lstm.d_Ji_ga,
        state.lstm.d_mc_ga, state.lstm.d_Jc_ga, state.lstm.d_mo_ga,
        state.lstm.d_Jo_ga, state.lstm.d_mc_prev, state.lstm.d_mca,
        state.lstm.d_Jca, d_state.d_delta_m, d_state.d_delta_S, z_pos_o,
        z_pos_o_lstm, b_pos_f, b_pos_i, b_pos_c, b_pos_o, no, net.input_seq_len,
        net.batch_size, d_theta.d_delta_mb, d_theta.d_delta_Sb);
}
